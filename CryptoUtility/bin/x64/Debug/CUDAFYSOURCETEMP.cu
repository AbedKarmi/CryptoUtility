
#include <hip/hip_runtime.h>

// RSA.CudafyEx
extern "C" __global__  void add( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0);

// RSA.CudafyEx
extern "C" __global__  void add( int* a, int aLen0,  int* b, int bLen0,  int* c, int cLen0)
{
	int x = blockIdx.x;
	bool flag = x < 10;
	if (flag)
	{
		c[(x)] = a[(x)] + b[(x)];
	}
}
